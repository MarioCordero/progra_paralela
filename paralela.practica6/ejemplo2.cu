#include <stdlib.h> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 
#include <math.h> 
#include <time.h> 
#include <hiprand/hiprand_kernel.h> 
 
#define TRIALS_PER_THREAD 4096 
#define BLOCKS 256 
#define THREADS 256 
// known value of pi
#define PI 3.1415926535
 
// With "__global__" the function will be called from the host but executed on the GPU, only works for this function
__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {

    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x; 
    int points_in_circle = 0; 
    float x, y; 

    // Initialize CURAND
    // 1234 es la semilla, tid es el id del hilo, 0 es el offset
    hiprand_init(1234, tid, 0, &states[tid]);

    for(int i = 0; i < TRIALS_PER_THREAD; i++) { 

        // -[x]
        // Generar valores aleatorios uniformes para x, y 
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);

        /*
        
        -curand_uniform:
        This is a function from CURAND (CUDA Random Number Generation library) that 
        generates a uniformly distributed floating-point random number between 0 
        (inclusive) and 1 (exclusive).

        -&states[tid]:
        states is an array of CURAND states. Each thread has its own CURAND state 
        to maintain independent random number sequences. tid is the identifier of 
        the current thread, so &states[tid] accesses the specific state of the current thread.

        */

        points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle. 
    }

    estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; // return estimate of pi 
}
 
float host_monte_carlo(long trials) { 
    float x, y; 
    long points_in_circle = 0; // Inicializar correctamente points_in_circle
    for(long i = 0; i < trials; i++) {
        x = rand() / (float) RAND_MAX; 
        y = rand() / (float) RAND_MAX; 
        points_in_circle += (x*x + y*y <= 1.0f); 
    } 
    return 4.0f * points_in_circle / trials;
} 
 
int main (int argc, char *argv[]) {

    clock_t start, stop; 
    float host[BLOCKS * THREADS]; 
    float *dev; 
    hiprandState *devStates; 
 
    printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", TRIALS_PER_THREAD, BLOCKS, THREADS); 
 
    start = clock(); 
 
    // -[x]
    // use cudaMalloc para pedir un bloque de memoria de tamaño BLOCKS * THREADS * sizeof(float). Guárdelo en dev. 
    hipMalloc((void **)&dev, BLOCKS * THREADS * sizeof(float));

    /*
    

    -cudaMalloc:
    A function from the CUDA API used to allocate memory on the GPU (device). It is analogous to the 
    malloc function in C, but in this case, the memory is allocated in the GPU's memory space.

    -**(void )&dev:
    dev is a pointer to a variable that will store the address of the memory allocated on the GPU.
    (void **)&dev casts the pointer &dev to a void** type, which is the type required by cudaMalloc 
    for the first argument. This is because cudaMalloc needs a double pointer (a pointer to a pointer) 
    to store the address of the allocated memory.

    -BLOCKS * THREADS * sizeof(float):

    BLOCKS is the number of execution blocks.
    THREADS is the number of threads per block.
    sizeof(float) returns the size in bytes of a float type.
    BLOCKS * THREADS is the total number of threads that will be launched.
    BLOCKS * THREADS * sizeof(float) calculates the total amount of memory in bytes needed to store a 
    float value for each thread. In this case, enough memory is being allocated to store a float value 
    for each thread in all the blocks.

    */
    
    // -[x]
    // use cudaMalloc para pedir un bloque de memoria de tamaño THREADS * BLOCKS * sizeof(curandState). Será usado para guardar el estado del generador de números aleatorios. Guardelo en devStates. 
    hipMalloc((void **)&devStates, BLOCKS * THREADS * sizeof(hiprandState));
    
    gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates); 

    // Use cudaMemcpy para copiar los datos de dev a host. 
    hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost);

    float pi_gpu = 0; // Inicializar correctamente pi_gpu

    for(int i = 0; i < BLOCKS * THREADS; i++) { 
        pi_gpu += host[i]; 
    } 
 
    pi_gpu /= (BLOCKS * THREADS); 
 
    stop = clock(); 
 
    printf("\nGPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC); 
 
    start = clock(); 

    float pi_cpu = host_monte_carlo(BLOCKS * THREADS * TRIALS_PER_THREAD); 

    stop = clock();

    printf("\nCPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC); 
 
    printf("\nCUDA estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI); 

    printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, pi_cpu - PI); 
  
    // Free memory
    hipFree(dev);
    hipFree(devStates);

    return 0; 
}